#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hipfft/hipfft.h>
#include<time.h>

#define PI 3.141592653589793
#define running_block_size 128



int *k_gas;
int *na, *nb, *nc;
int *na_device, *nb_device, *nc_device;
double *la_expanded, *lb_expanded, *lc_expanded;

double *grand_pontential, *grand_pontential_device;
double *Vext_device;
double *chem_device;
double *rho_bulk_device;

double *sqrt_dens_device;
double *gradient_device;
double *diameter_HS_device;
int *k_gas_device;
double *w_n0_FFT_device;
double *w_n3_FFT_device;
double *w_nv2x_FFT_device;
double *w_nv2y_FFT_device;
double *w_nv2z_FFT_device;
double *uatt_device;



hipfftHandle fft_plan_device;
hipfftDoubleComplex *FFT_in_device;
hipfftDoubleComplex *n0_FFT_in_device, *n1_FFT_in_device, *n2_FFT_in_device, *n3_FFT_in_device;
hipfftDoubleComplex *nv1x_FFT_in_device, *nv1y_FFT_in_device, *nv1z_FFT_in_device;
hipfftDoubleComplex *nv2x_FFT_in_device, *nv2y_FFT_in_device, *nv2z_FFT_in_device;





hipfftDoubleComplex *FFT_out_device;

hipfftDoubleComplex *n0_FFT_out_device, *n1_FFT_out_device, *n2_FFT_out_device, *n3_FFT_out_device;
hipfftDoubleComplex *nv1x_FFT_out_device, *nv1y_FFT_out_device, *nv1z_FFT_out_device;
hipfftDoubleComplex *nv2x_FFT_out_device, *nv2y_FFT_out_device, *nv2z_FFT_out_device;


double *denskr_device, *denski_device;





// __global__ defines the funciton that can be called from the host (CPU) and executed in the device (GPU)
__global__
void check_int(int n, int *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	for (j=0; j<n; j++)
	{
		for (i=index; i<n; i+=stride)
		{
			if (i==j)
			{
				//printf("index: %d\t%d\n", i, x[i]);
				printf("%d %d\n", i, x[i]);
			}
		}
	}
}

__global__
void check_double(int n, double *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	for (j=0; j<n; j++)
	{
		for (i=index; i<n; i+=stride)
		{
			if (i==j)
			{
				printf("index: %d\t%lf\n", i, x[i]);
			}
		}
	}
}




__global__
void check_complex(int n, hipfftDoubleComplex *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	for (j=0; j<n; j++)
	{
		for (i=index; i<n; i+=stride)
		{
			if (i==j)
			{
				printf("index: %d\t%lf\t%lf\n", i, x[i].x, x[i].y);
			}
		}
	}
}



























__global__
void cal_weight_density(int *na_device, int *nb_device, int *nc_device, 
	double *la_expanded_device, double *lb_expanded_device, double *lc_expanded_device,
	int *k_gas_device, double *diameter_HS_device,
	double *w_n0_FFT_device, double *w_n3_FFT_device, double *w_nv2x_FFT_device, double *w_nv2y_FFT_device, double *w_nv2z_FFT_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int index_gas, index_x, index_y, index_z;
	double fft_xk,  fft_yk, fft_zk;
	double dx, dy, dz;
	double fft_k;
	double fft_abs_kR;

	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]*k_gas_device[0]); i+=stride)
	{
		dx = 1.0*la_expanded_device[0]/na_device[0];
		dy = 1.0*lb_expanded_device[0]/nb_device[0];
		dz = 1.0*lc_expanded_device[0]/nc_device[0];

		// decompose the row, column information from the overal index
		index_gas = (int) ( (i)/(na_device[0]*nb_device[0]*nc_device[0]) );
		index_x = (int) ( (i-index_gas*na_device[0]*nb_device[0]*nc_device[0])/(nb_device[0]*nc_device[0]) );
		index_y = (int) ( (i-index_gas*na_device[0]*nb_device[0]*nc_device[0]-index_x*nb_device[0]*nc_device[0])/(nc_device[0]) );
		index_z = (int) (i-index_gas*na_device[0]*nb_device[0]*nc_device[0]-index_x*nb_device[0]*nc_device[0]-index_y*nc_device[0]);

		// the first point (origin) is unique
		if ((i%(na_device[0]*nb_device[0]*nc_device[0])) == 0)
		{
			w_n0_FFT_device[i] = 1.0 / (na_device[0]*nb_device[0]*nc_device[0]);
			w_n3_FFT_device[i] = 1.0*PI*pow(diameter_HS_device[index_gas],3)/6/(na_device[0]*nb_device[0]*nc_device[0]);
			w_nv2x_FFT_device[i] = 0;
			w_nv2y_FFT_device[i] = 0;
			w_nv2z_FFT_device[i] = 0;
		}
		else
		{
			// reformat the index into fft space with fftw fashion
			if (index_x<=0.5*na_device[0])
			{
				fft_xk = 2.0*PI*index_x/na_device[0]/dx;
			}
			else
			{
				fft_xk = -2.0*PI*(na_device[0]-index_x)/na_device[0]/dx;
			}
			if (index_y<=0.5*nb_device[0])
			{
				fft_yk = 2.0*PI*index_y/nb_device[0]/dy;
			}
			else
			{
				fft_yk = -2.0*PI*(nb_device[0]-index_y)/nb_device[0]/dy;
			}
			if (index_z<=0.5*nc_device[0])
			{
				fft_zk = 2.0*PI*index_z/nc_device[0]/dz;
			}
			else
			{
				fft_zk = -2.0*PI*(nc_device[0]-index_z)/nc_device[0]/dz;
			}

			fft_k = sqrt(pow(fft_xk,2)+pow(fft_yk,2)+pow(fft_zk,2));
			fft_abs_kR = 1.0*fabs(fft_k)*diameter_HS_device[index_gas]/2.0;

			w_n0_FFT_device[i] = sin(fft_abs_kR)/fft_abs_kR / (na_device[0]*nb_device[0]*nc_device[0]);
			w_n3_FFT_device[i] = 4.0*PI*(sin(fft_abs_kR)-fft_abs_kR*cos(fft_abs_kR))/pow(fabs(fft_k), 3) / 
			(na_device[0]*nb_device[0]*nc_device[0]);
			w_nv2x_FFT_device[i] = -1.0*w_n3_FFT_device[i]*fft_xk;
			w_nv2y_FFT_device[i] = -1.0*w_n3_FFT_device[i]*fft_yk;
			w_nv2z_FFT_device[i] = -1.0*w_n3_FFT_device[i]*fft_zk;
		}
	}
}












__global__
void cal_Vext(int *na_device, int *nb_device, int *nc_device, 
	double *la_expanded_device, double *lb_expanded_device, double *lc_expanded_device, 
	int *N_atoms_expanded_device, double *epsilon_host_star_expanded_device, double *sigma_host_expanded_device,
	double *x_host_expanded_device, double *y_host_expanded_device, double *z_host_expanded_device,
	int *k_gas_device, double *epsilon_star_device, double *sigma_device, double *rho_bulk_device, 
	double *diameter_HS_device, double *sqrt_dens_device, double *cutoff_device, double *Vext_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, ii;
	int index_gas, index_x, index_y, index_z;
	double pos_x, pos_y, pos_z;
	double dis_x, dis_y, dis_z;
	double dis;
	double epsilon_mix, sigma_mix;
	double pot, pot_cutoff;
	double dens;
	// printf("%d\n", na_device[0]*nb_device[0]*nc_device[0]*k_gas_device[0]);
	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]*k_gas_device[0]); i+=stride)
	{
		// decompose the row, column information from the overal index
		index_gas = (int) ( (i)/(na_device[0]*nb_device[0]*nc_device[0]) );
		index_x = (int) ( (i-index_gas*na_device[0]*nb_device[0]*nc_device[0])/(nb_device[0]*nc_device[0]) );
		index_y = (int) ( (i-index_gas*na_device[0]*nb_device[0]*nc_device[0]-index_x*nb_device[0]*nc_device[0])/(nc_device[0]) );
		index_z = (int) (i-index_gas*na_device[0]*nb_device[0]*nc_device[0]-index_x*nb_device[0]*nc_device[0]-index_y*nc_device[0]);

		Vext_device[i] = 0;

		pos_x = 1.0*index_x*la_expanded_device[0]/na_device[0];
		pos_y = 1.0*index_y*lb_expanded_device[0]/nb_device[0];
		pos_z = 1.0*index_z*lc_expanded_device[0]/nc_device[0];

		for (ii=0; ii<N_atoms_expanded_device[0]; ii++)
		{
			dis_x = pos_x - x_host_expanded_device[ii];
			dis_y = pos_y - y_host_expanded_device[ii];
			dis_z = pos_z - z_host_expanded_device[ii];
			if (dis_x > (0.5*la_expanded_device[0]))
			{
				dis_x = dis_x - la_expanded_device[0];
			}
			else if (dis_x < (-0.5*la_expanded_device[0]))
			{
				dis_x = dis_x + la_expanded_device[0];
			}

			if (dis_y > (0.5*lb_expanded_device[0]))
			{
				dis_y = dis_y - lb_expanded_device[0];
			}
			else if (dis_y < (-0.5*lb_expanded_device[0]))
			{
				dis_y = dis_y + lb_expanded_device[0];
			}

			if (dis_z > (0.5*lc_expanded_device[0]))
			{
				dis_z = dis_z - lc_expanded_device[0];
			}
			else if (dis_z < (-0.5*lc_expanded_device[0]))
			{
				dis_z = dis_z + lc_expanded_device[0];
			}
			dis = sqrt(pow(dis_x,2)+pow(dis_y,2)+pow(dis_z,2));
			if (dis<cutoff_device[0])
			{
				sigma_mix = 1.0*(sigma_device[index_gas]+sigma_host_expanded_device[ii])/2.0;
				epsilon_mix = sqrt(epsilon_star_device[index_gas]*epsilon_host_star_expanded_device[ii]);
				if (dis < 0.1*sigma_mix)
				{
					dis = 0.1*sigma_mix;
				}
				pot = 4.0*epsilon_mix*(pow((1.0*sigma_mix/dis),12)-pow((1.0*sigma_mix/dis),6));
				pot_cutoff = 4.0*epsilon_mix*(pow((1.0*sigma_mix/cutoff_device[0]),12)-pow((1.0*sigma_mix/cutoff_device[0]),6));
				Vext_device[i] = Vext_device[i] + (pot - pot_cutoff);
			}
		}

		dens = rho_bulk_device[index_gas]*exp(-Vext_device[i])/100;
		// dens = rho_bulk_device[index_gas]/100;


		if (dens > 2.0/pow(diameter_HS_device[index_gas],3)/k_gas_device[0])
		{
			dens = 2.0/pow(diameter_HS_device[index_gas],3)/k_gas_device[0];
		}

		sqrt_dens_device[i] = sqrt(dens);
	}
}











__global__
void ini_device_value_double(int n, double *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	for (i=index; i<n; i+=stride)
	{
		x[i] = 0;
	}
}



__global__
void ini_device_value_complex(int n, hipfftDoubleComplex *x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	for (i=index; i<n; i+=stride)
	{
		x[i].x = 0;
		x[i].y = 0;
	}
}



__global__
void real_split_into_complex_batch(int n, double *in, hipfftDoubleComplex *out, int num_batch)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int index_in;

	for (i=index; i<n; i+=stride)
	{
		index_in = num_batch*n + i;
		out[i].x = in[index_in]*in[index_in];
		out[i].y = 0;
	}
}

__global__
void real_split_into_complex_separate_batch(int n, double *in_r, double *in_i, hipfftDoubleComplex *out, int num_batch)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int index_in;

	for (i=index; i<n; i+=stride)
	{
		index_in = num_batch*n + i;
		out[i].x = in_r[index_in];
		out[i].y = in_i[index_in];
	}
}

__global__
void complext_split_into_real_separate_batch(int n, double *out_r, double *out_i, hipfftDoubleComplex *in, int num_batch)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int index_in;

	for (i=index; i<n; i+=stride)
	{
		index_in = num_batch*n + i;
		out_r[index_in] = in[i].x;
		out_i[index_in] = in[i].y;
	}
}






__global__
void add_weighted_density(int *na_device, int *nb_device, int *nc_device, double *diameter_HS_device, int *k_gas_device, 
	hipfftDoubleComplex *FFT_out_device, hipfftDoubleComplex *n0_FFT_in_device, hipfftDoubleComplex *n1_FFT_in_device, 
	hipfftDoubleComplex *n2_FFT_in_device, hipfftDoubleComplex *n3_FFT_in_device, 
	hipfftDoubleComplex *nv1x_FFT_in_device, hipfftDoubleComplex *nv1y_FFT_in_device, hipfftDoubleComplex *nv1z_FFT_in_device, 
	hipfftDoubleComplex *nv2x_FFT_in_device, hipfftDoubleComplex *nv2y_FFT_in_device, hipfftDoubleComplex *nv2z_FFT_in_device, 
	double *w_n0_FFT_device, double *w_n3_FFT_device, double *w_nv2x_FFT_device, double *w_nv2y_FFT_device, double *w_nv2z_FFT_device, 
	double *uatt_device, double *denskr_device, double  *denski_device,	int j)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, jj;
	int batch_i;
	int temp_index1, temp_index2, temp_index;
	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		batch_i = i + j*na_device[0]*nb_device[0]*nc_device[0];

		n0_FFT_in_device[i].x = n0_FFT_in_device[i].x + FFT_out_device[i].x*w_n0_FFT_device[batch_i];
		n0_FFT_in_device[i].y = n0_FFT_in_device[i].y + FFT_out_device[i].y*w_n0_FFT_device[batch_i];
		n1_FFT_in_device[i].x = n1_FFT_in_device[i].x + FFT_out_device[i].x*w_n0_FFT_device[batch_i]*diameter_HS_device[j]*0.5;
		n1_FFT_in_device[i].y = n1_FFT_in_device[i].y + FFT_out_device[i].y*w_n0_FFT_device[batch_i]*diameter_HS_device[j]*0.5;
		n2_FFT_in_device[i].x = n2_FFT_in_device[i].x + FFT_out_device[i].x*w_n0_FFT_device[batch_i]*PI*diameter_HS_device[j]*diameter_HS_device[j];
		n2_FFT_in_device[i].y = n2_FFT_in_device[i].y + FFT_out_device[i].y*w_n0_FFT_device[batch_i]*PI*diameter_HS_device[j]*diameter_HS_device[j];
		n3_FFT_in_device[i].x = n3_FFT_in_device[i].x + FFT_out_device[i].x*w_n3_FFT_device[batch_i];
		n3_FFT_in_device[i].y = n3_FFT_in_device[i].y + FFT_out_device[i].y*w_n3_FFT_device[batch_i];

		nv1x_FFT_in_device[i].x = nv1x_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2x_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		nv1x_FFT_in_device[i].y = nv1x_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2x_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		nv1y_FFT_in_device[i].x = nv1y_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2y_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		nv1y_FFT_in_device[i].y = nv1y_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2y_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		nv1z_FFT_in_device[i].x = nv1z_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2z_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		nv1z_FFT_in_device[i].y = nv1z_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2z_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);

		nv2x_FFT_in_device[i].x = nv2x_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2x_FFT_device[batch_i];
		nv2x_FFT_in_device[i].y = nv2x_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2x_FFT_device[batch_i];
		nv2y_FFT_in_device[i].x = nv2y_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2y_FFT_device[batch_i];
		nv2y_FFT_in_device[i].y = nv2y_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2y_FFT_device[batch_i];
		nv2z_FFT_in_device[i].x = nv2z_FFT_in_device[i].x + -FFT_out_device[i].y*w_nv2z_FFT_device[batch_i];
		nv2z_FFT_in_device[i].y = nv2z_FFT_in_device[i].y + FFT_out_device[i].x*w_nv2z_FFT_device[batch_i];

		for (jj=0; jj<k_gas_device[0]; jj++)
		{
			if (jj>j)
			{
				temp_index1 = jj;
				temp_index2 = j;
			}
			else
			{
				temp_index1 = j;
				temp_index2 = jj;
			}
			temp_index = (int) floor((temp_index1+0)*(temp_index1+1)*0.5)*na_device[0]*nb_device[0]*nc_device[0] 
			+ temp_index2*na_device[0]*nb_device[0]*nc_device[0] + i;

			denskr_device[i+jj*na_device[0]*nb_device[0]*nc_device[0]] += FFT_out_device[i].x*uatt_device[temp_index];
			denski_device[i+jj*na_device[0]*nb_device[0]*nc_device[0]] += FFT_out_device[i].y*uatt_device[temp_index];
		}
	}
}





__global__
void cal_F_deri(int *na_device, int *nb_device, int *nc_device, 
	hipfftDoubleComplex *n0_FFT_out_device,
	hipfftDoubleComplex *n1_FFT_out_device, hipfftDoubleComplex *n2_FFT_out_device, hipfftDoubleComplex *n3_FFT_out_device, 
	hipfftDoubleComplex *nv1x_FFT_out_device, hipfftDoubleComplex *nv1y_FFT_out_device, hipfftDoubleComplex *nv1z_FFT_out_device, 
	hipfftDoubleComplex *nv2x_FFT_out_device, hipfftDoubleComplex *nv2y_FFT_out_device, hipfftDoubleComplex *nv2z_FFT_out_device,
	hipfftDoubleComplex *n0_FFT_in_device,
	hipfftDoubleComplex *n1_FFT_in_device, hipfftDoubleComplex *n2_FFT_in_device, hipfftDoubleComplex *n3_FFT_in_device, 
	hipfftDoubleComplex *nv1x_FFT_in_device, hipfftDoubleComplex *nv1y_FFT_in_device, hipfftDoubleComplex *nv1z_FFT_in_device, 
	hipfftDoubleComplex *nv2x_FFT_in_device, hipfftDoubleComplex *nv2y_FFT_in_device, hipfftDoubleComplex *nv2z_FFT_in_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	double tn0, tn1, tn2, tn3;
	// double tnv1x, tnv1y, tnv1z;
	double tnv2x, tnv2y, tnv2z;
	double tn30, tn31, tn32, tn33, tnv21, tnv22, t2n3, t3n3;

	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		tn0 = n0_FFT_out_device[i].x;
		tn1 = n1_FFT_out_device[i].x;
		tn2 = n2_FFT_out_device[i].x;
		tn3 = n3_FFT_out_device[i].x;

		if (tn0<0)
		{
			tn0 = 0;
			// if (tn0<-0.0001)
			// {
			// 	// printf("Fatal Error!!!!!!! tn0!!!!!!!\n");
			// }
		}

		if (tn3<0)
		{
			tn3 = 0;
			// if (tn3<-0.0001)
			// {
			// 	// printf("Fatal Error!!!!!!! tn3!!!!!!!\n");
			// }
		}
		else if (tn3>0.99)
		{
			tn3 = 0.99;
			// printf("Fatal Error!!!!!!! tn3!!!!!!!\n");
		}

		tnv2x = nv2x_FFT_out_device[i].x;
		tnv2y = nv2y_FFT_out_device[i].x;
		tnv2z = nv2z_FFT_out_device[i].x;

		if (tn3<1e-5)
		{
			n0_FFT_in_device[i].x = 0;
			n0_FFT_in_device[i].y = 0;
			n1_FFT_in_device[i].x = 0;
			n1_FFT_in_device[i].y = 0;
			n2_FFT_in_device[i].x = 0;
			n2_FFT_in_device[i].y = 0;
			n3_FFT_in_device[i].x = 0;
			n3_FFT_in_device[i].y = 0;

			nv1x_FFT_in_device[i].x = 0;
			nv1x_FFT_in_device[i].y = 0;
			nv1y_FFT_in_device[i].x = 0;
			nv1y_FFT_in_device[i].y = 0;
			nv1z_FFT_in_device[i].x = 0;
			nv1z_FFT_in_device[i].y = 0;

			nv2x_FFT_in_device[i].x = 0;
			nv2x_FFT_in_device[i].y = 0;
			nv2y_FFT_in_device[i].x = 0;
			nv2y_FFT_in_device[i].y = 0;
			nv2z_FFT_in_device[i].x = 0;
			nv2z_FFT_in_device[i].y = 0;
		}
		else
		{
			tn31 = 1-tn3;
            tn30 = log(tn31);
            tn31 = 1.0/tn31;
            tn32 = tn31*tn31;
            tn33 = tn31*tn32;
            tnv22 = tnv2x*tnv2x + tnv2y*tnv2y + tnv2z*tnv2z;
            tnv21 = tnv2x*nv1x_FFT_out_device[i].x + tnv2y*nv1y_FFT_out_device[i].x + tnv2z*nv1z_FFT_out_device[i].x;
            t2n3 = 1.0/tn3/tn3;
            t3n3 = t2n3/tn3;
            // printf("%.5e\t%.5e\n", nv1y_FFT_out_device[i].x, (1.0/(1-tn3)));
            
            // nv2y_FFT_in_device[i].x = -1.0*(tn30/tn3+tn32)*tn2*tnv2y/(6.0*PI*tn3) - tnv1y*tn31;


            n0_FFT_in_device[i].x = -tn30;
			n0_FFT_in_device[i].y = 0;
			n1_FFT_in_device[i].x = tn2*tn31;
			n1_FFT_in_device[i].y = 0;
			n2_FFT_in_device[i].x = tn1*tn31 + 1.0*(1.0*tn30/tn3+tn32)*(tn2*tn2-tnv22)/(12.0*PI*tn3);
			n2_FFT_in_device[i].y = 0;
			n3_FFT_in_device[i].x = ((1.0*tn30*t3n3/(18.0*PI))+(1.0-3*tn3+1.0/tn32)*t2n3*tn33/(36.0*PI))*(3*tn2*tnv22-tn2*tn2*tn2)
	            					+ tn0*tn31 + (tn1*tn2-tnv21)*tn32 + 80.0*exp((tn3-0.7)*80.0);
			n3_FFT_in_device[i].y = 0;

			nv1x_FFT_in_device[i].x = -tnv2x*tn31;
			nv1x_FFT_in_device[i].y = 0;
			nv1y_FFT_in_device[i].x = -tnv2y*tn31;
			nv1y_FFT_in_device[i].y = 0;
			nv1z_FFT_in_device[i].x = -tnv2z*tn31;
			nv1z_FFT_in_device[i].y = 0;

			nv2x_FFT_in_device[i].x = -1.0*(tn30/tn3+tn32)*tn2*tnv2x/(6.0*PI*tn3) - nv1x_FFT_out_device[i].x*tn31;
			nv2x_FFT_in_device[i].y = 0;
			nv2y_FFT_in_device[i].x = -1.0*(log(1-tn3)/tn3+tn32)*tn2*tnv2y/(6.0*PI*tn3) - nv1y_FFT_out_device[i].x*(1.0/(1-n3_FFT_out_device[i].x));
			// nv2y_FFT_in_device[i].x = -1.0*(tn30/tn3+tn32)*tn2*tnv2y/(6.0*PI*tn3) - tnv1y*tn31;
			nv2y_FFT_in_device[i].y = 0;
			nv2z_FFT_in_device[i].x = -1.0*(log(1-tn3)/tn3+tn32)*tn2*tnv2z/(6.0*PI*tn3) - nv1z_FFT_out_device[i].x*(1.0/(1-n3_FFT_out_device[i].x));
			// nv2z_FFT_in_device[i].x = -1.0*(tn30/tn3+tn32)*tn2*tnv2z/(6.0*PI*tn3) - tnv1z*tn31;
			nv2z_FFT_in_device[i].y = 0;
		}
	}
}





__global__
void sum_F_deri(int *na_device, int *nb_device, int *nc_device, double *diameter_HS_device, int *k_gas_device, 
	hipfftDoubleComplex *FFT_in_device, hipfftDoubleComplex *n0_FFT_out_device, 
	hipfftDoubleComplex *n1_FFT_out_device, hipfftDoubleComplex *n2_FFT_out_device, hipfftDoubleComplex *n3_FFT_out_device, 
	hipfftDoubleComplex *nv1x_FFT_out_device, hipfftDoubleComplex *nv1y_FFT_out_device, hipfftDoubleComplex *nv1z_FFT_out_device, 
	hipfftDoubleComplex *nv2x_FFT_out_device, hipfftDoubleComplex *nv2y_FFT_out_device, hipfftDoubleComplex *nv2z_FFT_out_device, 
	double *w_n0_FFT_device, double *w_n3_FFT_device, double *w_nv2x_FFT_device, double *w_nv2y_FFT_device, double *w_nv2z_FFT_device, 
	int j)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int batch_i;
	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		batch_i = i + j*na_device[0]*nb_device[0]*nc_device[0];

		FFT_in_device[i].x = n0_FFT_out_device[i].x*w_n0_FFT_device[batch_i];
		FFT_in_device[i].y = n0_FFT_out_device[i].y*w_n0_FFT_device[batch_i];

		FFT_in_device[i].x += n1_FFT_out_device[i].x*w_n0_FFT_device[batch_i]*diameter_HS_device[j]*0.5;
		FFT_in_device[i].y += n1_FFT_out_device[i].y*w_n0_FFT_device[batch_i]*diameter_HS_device[j]*0.5;

		FFT_in_device[i].x += n2_FFT_out_device[i].x*w_n0_FFT_device[batch_i]*PI*diameter_HS_device[j]*diameter_HS_device[j];
		FFT_in_device[i].y += n2_FFT_out_device[i].y*w_n0_FFT_device[batch_i]*PI*diameter_HS_device[j]*diameter_HS_device[j];

		FFT_in_device[i].x += n3_FFT_out_device[i].x*w_n3_FFT_device[batch_i];
		FFT_in_device[i].y += n3_FFT_out_device[i].y*w_n3_FFT_device[batch_i];



		FFT_in_device[i].x += nv1x_FFT_out_device[i].y*w_nv2x_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		FFT_in_device[i].y += -nv1x_FFT_out_device[i].x*w_nv2x_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		FFT_in_device[i].x += nv1y_FFT_out_device[i].y*w_nv2y_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		FFT_in_device[i].y += -nv1y_FFT_out_device[i].x*w_nv2y_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		FFT_in_device[i].x += nv1z_FFT_out_device[i].y*w_nv2z_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);
		FFT_in_device[i].y += -nv1z_FFT_out_device[i].x*w_nv2z_FFT_device[batch_i]*0.5/(PI*diameter_HS_device[j]);



		FFT_in_device[i].x += nv2x_FFT_out_device[i].y*w_nv2x_FFT_device[batch_i];
		FFT_in_device[i].y += -nv2x_FFT_out_device[i].x*w_nv2x_FFT_device[batch_i];
		FFT_in_device[i].x += nv2y_FFT_out_device[i].y*w_nv2y_FFT_device[batch_i];
		FFT_in_device[i].y += -nv2y_FFT_out_device[i].x*w_nv2y_FFT_device[batch_i];
		FFT_in_device[i].x += nv2z_FFT_out_device[i].y*w_nv2z_FFT_device[batch_i];
		FFT_in_device[i].y += -nv2z_FFT_out_device[i].x*w_nv2z_FFT_device[batch_i];
	}
}





void new_MBWR(int *k_gas, double *epsilon, double *sigma, double *temperature, double *rho_bulk, double *chem, double *bulk_pressure)
{
    int i, ii;

    double *x, *a, *b, *c, *d, *G, *gamma, *F;
    x = (double *) malloc(sizeof(double)*32);
    a = (double *) malloc(sizeof(double)*8);
    b = (double *) malloc(sizeof(double)*6);
    c = (double *) malloc(sizeof(double)*8);
    d = (double *) malloc(sizeof(double)*6);
    G = (double *) malloc(sizeof(double)*6);
    gamma = (double *) malloc(sizeof(double));
    F = (double *) malloc(sizeof(double));
    double *sum_a, *sum_b;
    sum_a = (double *) malloc(sizeof(double));
    sum_b = (double *) malloc(sizeof(double));

    double *rho_r, *T_r;
    rho_r = (double *) malloc(sizeof(double));
    T_r = (double *) malloc(sizeof(double));

    double *F_r, *P_r, *U_r;
    F_r = (double *) malloc(sizeof(double));
    P_r = (double *) malloc(sizeof(double));
    U_r = (double *) malloc(sizeof(double));


    // mixing LJ parameter for mixture
    double *epsilon_mix, *sigma_mix;
    double *tot_rho;
    double *sigma_x3;
    double *epsilon_x;
    epsilon_mix = (double *) malloc(sizeof(double));
    sigma_mix = (double *) malloc(sizeof(double));
    tot_rho = (double *) malloc(sizeof(double));
    sigma_x3 = (double *) malloc(sizeof(double));
    epsilon_x = (double *) malloc(sizeof(double));
    sigma_x3[0] = 0;
    epsilon_x[0] = 0;
    tot_rho[0] = 0;
    for (i=0; i<k_gas[0]; i++)
    {
        tot_rho[0] = tot_rho[0] + rho_bulk[i];
        for (ii=0; ii<k_gas[0]; ii++)
        {
            epsilon_mix[0] = sqrt(epsilon[i]*epsilon[ii]);
            sigma_mix[0] = 0.5*(sigma[i]+sigma[ii]);
            sigma_x3[0] = sigma_x3[0] + rho_bulk[i]*rho_bulk[ii]*pow(sigma_mix[0],3);
            epsilon_x[0] = epsilon_x[0] + rho_bulk[i]*rho_bulk[ii]*epsilon_mix[0]*pow(sigma_mix[0],3);
        }
    }
    sigma_x3[0] = 1.0*sigma_x3[0]/tot_rho[0]/tot_rho[0];
    epsilon_x[0] = 1.0*epsilon_x[0]/tot_rho[0]/tot_rho[0]/sigma_x3[0];

    rho_r[0] = sigma_x3[0]*tot_rho[0];
    T_r[0] = 1.0*temperature[0]/epsilon_x[0];

    // MBWR parameter from Keith E. Gubbins
    x[0] = 0.8623085097507421;
    x[1] = 2.976218765822098;
    x[2] = -8.402230115796038;
    x[3] = 0.1054136629203555;
    x[4] = -0.8564583828174598;
    x[5] = 1.582759470107601;
    x[6] = 0.7639421948305453;
    x[7] = 1.753173414312048;
    x[8] = 2.798291772190376e+03;
    x[9] = -4.8394220260857657e-02;
    x[10] = 0.9963265197721935;
    x[11] = -3.698000291272493e+01;
    x[12] = 2.084012299434647e+01;
    x[13] = 8.305402124717285e+01;
    x[14] = -9.574799715203068e+02;
    x[15] = -1.477746229234994e+02;
    x[16] = 6.398607852471505e+01;
    x[17] = 1.603993673294834e+01;
    x[18] = 6.805916615864377e+01;
    x[19] = -2.791293578795945e+03;
    x[20] = -6.245128304568454;
    x[21] = -8.116836104958410e+03;
    x[22] = 1.488735559561229e+01;
    x[23] = -1.059346754655084e+04;
    x[24] = -1.131607632802822e+02;
    x[25] = -8.867771540418822e+03;
    x[26] = -3.986982844450543e+01;
    x[27] = -4.689270299917261e+03;
    x[28] = 2.593535277438717e+02;
    x[29] = -2.694523589434903e+03;
    x[30] = -7.218487631550215e+02;
    x[31] = 1.721802063863269e+02;
    // MBWR parameters of a
    a[0] = x[0]*T_r[0] + x[1]*pow(T_r[0], 0.5) + x[2] + x[3]/T_r[0] + x[4]/pow(T_r[0], 2);
    a[1] = x[5]*T_r[0] + x[6] + x[7]/T_r[0] + x[8]/pow(T_r[0], 2);
    a[2] = x[9]*T_r[0] + x[10] + x[11]/T_r[0];
    a[3] = x[12];
    a[4] = x[13]/T_r[0] + x[14]/pow(T_r[0], 2);
    a[5] = x[15]/T_r[0];
    a[6] = x[16]/T_r[0] + x[17]/pow(T_r[0], 2);
    a[7] = x[18]/pow(T_r[0], 2);
    // MBWR parameters of b
    b[0] = x[19]/pow(T_r[0], 2) + x[20]/pow(T_r[0], 3);
    b[1] = x[21]/pow(T_r[0], 2) + x[22]/pow(T_r[0], 4);
    b[2] = x[23]/pow(T_r[0], 2) + x[24]/pow(T_r[0], 3);
    b[3] = x[25]/pow(T_r[0], 2) + x[26]/pow(T_r[0], 4);
    b[4] = x[27]/pow(T_r[0], 2) + x[28]/pow(T_r[0], 3);
    b[5] = x[29]/pow(T_r[0], 2) + x[30]/pow(T_r[0], 3) + x[31]/pow(T_r[0], 4);
    // MBWR parameters of c
    c[0] = x[1]*sqrt(T_r[0])*0.5 + x[2] + 2*x[3]/T_r[0] + 3*x[4]/pow(T_r[0],2);
    c[1] = x[6] + 2*x[7]/T_r[0] + 3*x[8]/pow(T_r[0],2);
    c[2] = x[10] + 2*x[11]/T_r[0];
    c[3] = x[12];
    c[4] = 2*x[13]/T_r[0] + 3*x[14]/pow(T_r[0],2);
    c[5] = 2*x[15]/T_r[0];
    c[6] = 2*x[16]/T_r[0] + 3*x[17]/pow(T_r[0],2);
    c[7] = 3*x[18]/pow(T_r[0],2);
    // MBWE parameters of d
    d[0] = 3*x[19]/pow(T_r[0],2) + 4*x[20]/pow(T_r[0],3);
    d[1] = 3*x[21]/pow(T_r[0],2) + 5*x[22]/pow(T_r[0],4);
    d[2] = 3*x[23]/pow(T_r[0],2) + 4*x[24]/pow(T_r[0],3);
    d[3] = 3*x[25]/pow(T_r[0],2) + 5*x[26]/pow(T_r[0],4);
    d[4] = 3*x[27]/pow(T_r[0],2) + 4*x[28]/pow(T_r[0],3);
    d[5] = 3*x[29]/pow(T_r[0],2) + 4*x[30]/pow(T_r[0],3) + 5*x[31]/pow(T_r[0],4);
    // MBWR parameter of G
    gamma[0] = 3;
    F[0] = exp(-gamma[0]*pow(rho_r[0],2));
    G[0] = (1-F[0])/(2*gamma[0]);
    for (i=1; i<6; i++)
    {
        G[i] = i/gamma[0]*G[i-1] - F[0]/(2*gamma[0])*pow(rho_r[0],2*i);
    }



    
    // calculate reduced residual Helmholtz Free energy
    sum_a[0] = 0;
    sum_b[0] = 0;
    for (i=0; i<=7; i++)
    {
        sum_a[0] = sum_a[0] + a[i]*pow(rho_r[0], (i+1))/(i+1);
    }
    for (i=0; i<=5; i++)
    {
        sum_b[0] = sum_b[0] + b[i]*G[i];
    }
    F_r[0] = sum_a[0] + sum_b[0];
    // calculate reduced reduced bulk pressure
    sum_a[0] = 0;
    sum_b[0] = 0;
    for (i=0; i<=7; i++)
    {
        sum_a[0] = sum_a[0] + a[i]*pow(rho_r[0], (i+2));
    }
    for (i=0; i<=5; i++)
    {
        sum_b[0] = sum_b[0] + F[0]*b[i]*pow(rho_r[0], (2*i+3));
    }
    P_r[0] = rho_r[0]*T_r[0] + sum_a[0] + sum_b[0];
    bulk_pressure[0] = P_r[0]*epsilon_x[0]*1.38064852*pow(10,-23)/(sigma_x3[0]*pow(10,-30))*pow(10,-5);
    // calculate reduced internal energy
    sum_a[0] = 0;
    sum_b[0] = 0;
    for (i=0; i<=7; i++)
    {
        sum_a[0] = sum_a[0] + c[i]*pow(rho_r[0], (i+1))/(i+1);
    }
    for (i=0; i<=5; i++)
    {
        sum_b[0] = sum_b[0] + d[i]*G[i];
    }
    U_r[0] = sum_a[0] + sum_b[0];


    // calculate the chemical potential for each species
    double *deriv1, *deriv2, *deriv3;
    deriv1 = (double *) malloc(sizeof(double));
    deriv2 = (double *) malloc(sizeof(double));
    deriv3 = (double *) malloc(sizeof(double));
    for (i=0; i<k_gas[0]; i++)
    {
        deriv3[0] = 0;
        for (ii=0; ii<k_gas[0]; ii++)
        {
            sigma_mix[0] = 0.5*(sigma[i]+sigma[ii]);
            deriv3[0] = deriv3[0] + rho_bulk[ii]*pow(sigma_mix[0],3);
        }
        deriv3[0] = deriv3[0]/tot_rho[0];
        deriv3[0] = deriv3[0] - sigma_x3[0];
        deriv3[0] = 2.0*deriv3[0]/tot_rho[0];

        deriv1[0] = 0;
        for (ii=0; ii<k_gas[0]; ii++)
        {
            epsilon_mix[0] = sqrt(epsilon[i]*epsilon[ii]);
            sigma_mix[0] = 0.5*(sigma[i]+sigma[ii]);
            deriv1[0] = deriv1[0] + rho_bulk[ii]*epsilon_mix[0]*pow(sigma_mix[0],3);
        }
        deriv1[0] = deriv1[0]/tot_rho[0]/sigma_x3[0] - epsilon_x[0];
        deriv1[0] = 2.0*deriv1[0]/tot_rho[0];
        deriv1[0] = -epsilon_x[0]*deriv3[0]/sigma_x3[0] + deriv1[0];

        deriv2[0] = (1.0*P_r[0]/pow(rho_r[0],2) - T_r[0]/rho_r[0]) * (sigma_x3[0]+tot_rho[0]*deriv3[0]);
        deriv2[0] = deriv2[0] - (F_r[0]-U_r[0])*deriv1[0]/epsilon_x[0];

        chem[i] = (F_r[0]*epsilon_x[0] + F_r[0]*tot_rho[0]*deriv1[0] + tot_rho[0]*epsilon_x[0]*deriv2[0])/temperature[0] + log(rho_bulk[i]);
        // chem[i] = (F_r[0]*epsilon_x[0] + F_r[0]*tot_rho[0]*deriv1[0] + tot_rho[0]*epsilon_x[0]*deriv2[0]) + log(rho_bulk[i]);
    }
}
























__global__
void cal_lj_pre_freq_batch(int *na_device, int *nb_device, int *nc_device, double *la_expanded_device, double *lb_expanded_device, 
	double *lc_expanded_device, double *sigma_device, double *epsilon_star_device, double *cutoff_device, int j, int jj, 
	hipfftDoubleComplex *FFT_in_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int index_x, index_y, index_z;
	double dx, dy, dz;
	double x, y, z;
	double sigma_mix, epsilon_mix;
	double dis;
	double pot_cutoff, pot;

	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		dx = 1.0*la_expanded_device[0]/na_device[0];
		dy = 1.0*lb_expanded_device[0]/nb_device[0];
		dz = 1.0*lc_expanded_device[0]/nc_device[0];

		// decompose the row, column information from the overall index
		index_x = (int) ( (i)/(nb_device[0]*nc_device[0]) );
		index_y = (int) ( (i-index_x*nb_device[0]*nc_device[0])/(nc_device[0]) );
		index_z = (int) ( i-index_x*nb_device[0]*nc_device[0]-index_y*nc_device[0] );

		sigma_mix = 0.5*(sigma_device[j]+sigma_device[jj]);
		epsilon_mix = sqrt((epsilon_star_device[j]*epsilon_star_device[jj]));

		pot_cutoff = 4.0*epsilon_mix* ( pow((1.0*sigma_mix/cutoff_device[0]),12) - pow((1.0*sigma_mix/cutoff_device[0]),6) );

		if (index_x<0.5*na_device[0])
		{
			x = index_x*dx;
		}
		else
		{
			x = (index_x-na_device[0])*dx;
		}
		if (index_y<0.5*nb_device[0])
		{
			y = index_y*dy;
		}
		else
		{
			y = (index_y-nb_device[0])*dy;
		}
		if (index_z<0.5*nc_device[0])
		{
			z = index_z*dz;
		}
		else
		{
			z = (index_z-nc_device[0])*dz;
		}

		dis = sqrt(pow(x,2)+pow(y,2)+pow(z,2));

		if (dis < sigma_mix)
		{
			pot = 0;
		}
		else if  (dis>cutoff_device[0])
		{
			pot = 0;
		}
		else
		{
			pot = 4.0*epsilon_mix*(pow((1.0*sigma_mix/dis),12)-pow((1.0*sigma_mix/dis),6)) - pot_cutoff;
		}

		FFT_in_device[i].x = pot;
		FFT_in_device[i].y = 0;
	}
}








__global__
void cal_lj_freq_store_batch(int *na_device, int *nb_device, int *nc_device, double *la_expanded_device, 
	double *lb_expanded_device, double *lc_expanded_device, int j, int jj, hipfftDoubleComplex *FFT_out_device, 
	double *uatt_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int nijk_gas;
	double dx, dy, dz;

	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		dx = 1.0*la_expanded_device[0]/na_device[0];
		dy = 1.0*lb_expanded_device[0]/nb_device[0];
		dz = 1.0*lc_expanded_device[0]/nc_device[0];

		nijk_gas = i + jj*na_device[0]*nb_device[0]*nc_device[0] + floor((j+0)*(j+1)*0.5)*na_device[0]*nb_device[0]*nc_device[0];
		uatt_device[nijk_gas] = 1.0*FFT_out_device[i].x*dx*dy*dz/(na_device[0]*nb_device[0]*nc_device[0]);
	}
}





__global__
void cal_grand_potential(int *na_device, int *nb_device, int *nc_device, int *k_gas_device, double *Vext_device, 
	double *chem_device,  double *sqrt_dens_device, double *rho_bulk_device, hipfftDoubleComplex *n0_FFT_out_device, 
	hipfftDoubleComplex *n1_FFT_out_device, hipfftDoubleComplex *n2_FFT_out_device, hipfftDoubleComplex *n3_FFT_out_device, 
	hipfftDoubleComplex *nv1x_FFT_out_device, hipfftDoubleComplex *nv1y_FFT_out_device, hipfftDoubleComplex *nv1z_FFT_out_device, 
	hipfftDoubleComplex *nv2x_FFT_out_device, hipfftDoubleComplex *nv2y_FFT_out_device, hipfftDoubleComplex *nv2z_FFT_out_device,
	double *denskr_device, double *grand_pontential_device)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	int nijk_gas;
	double tn0, tn1, tn2, tn3;
	double tnv1x, tnv1y, tnv1z;
	double tnv2x, tnv2y, tnv2z;
	double tn30, tn31, tn32, tn33, tnv21, tnv22, t2n3, t3n3;
	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		grand_pontential_device[i] = 0;
		for (j=0; j<k_gas_device[0]; j++)
		{
			nijk_gas = i + j*na_device[0]*nb_device[0]*nc_device[0];
			grand_pontential_device[i] += (Vext_device[nijk_gas]-chem_device[j])*sqrt_dens_device[nijk_gas]*sqrt_dens_device[nijk_gas];

			if ((sqrt_dens_device[nijk_gas]*sqrt_dens_device[nijk_gas])>1.0e-5*rho_bulk_device[j])
			{
				grand_pontential_device[i] += sqrt_dens_device[nijk_gas]*sqrt_dens_device[nijk_gas]*(log(sqrt_dens_device[nijk_gas]*sqrt_dens_device[nijk_gas])-1);
			}
		}

		tn0 = n0_FFT_out_device[i].x;
		tn1 = n1_FFT_out_device[i].x;
		tn2 = n2_FFT_out_device[i].x;
		tn3 = n3_FFT_out_device[i].x;

		if (tn0<0)
		{
			tn0 = 0;
			// if (tn0<-0.0001)
			// {
			// 	// printf("Fatal Error!!!!!!! tn0!!!!!!!\n");
			// }
		}

		if (tn3<0)
		{
			tn3 = 0;
			// if (tn3<-0.0001)
			// {
			// 	// printf("Fatal Error!!!!!!! tn3!!!!!!!\n");
			// }
		}
		else if (tn3>0.99)
		{
			tn3 = 0.99;
			// printf("Fatal Error!!!!!!! tn3!!!!!!!\n");
		}
		tnv1x = nv1x_FFT_out_device[i].x;
		tnv1y = nv1y_FFT_out_device[i].x;
		tnv1z = nv1z_FFT_out_device[i].x;
		tnv2x = nv2x_FFT_out_device[i].x;
		tnv2y = nv2y_FFT_out_device[i].x;
		tnv2z = nv2z_FFT_out_device[i].x;

		if (tn3>=1e-5)
		{
			tn31 = 1-tn3;
            tn30 = log(tn31);
            tn31 = 1.0/tn31;
            tn32 = tn31*tn31;
            tn33 = tn31*tn32;
            tnv22 = tnv2x*tnv2x + tnv2y*tnv2y + tnv2z*tnv2z;
            tnv21 = tnv2x*tnv1x + tnv2y*tnv1y + tnv2z*tnv1z;
            t2n3 = 1.0/tn3/tn3;
            t3n3 = t2n3/tn3;

            grand_pontential_device[i] += -tn0*tn30;

            grand_pontential_device[i] += (tn1*tn2-tnv21)*tn31;

            grand_pontential_device[i] += (pow(tn2,3)-3*tn2*tnv22)*(tn3+tn30/tn32)/(36.0*PI*tn3*tn3)*tn32;

            grand_pontential_device[i] += exp(80*(tn3-0.7));

            for (j=0; j<k_gas_device[0]; j++)
            {
                nijk_gas = i + j*na_device[0]*nb_device[0]*nc_device[0];
                grand_pontential_device[i] += 0.5*sqrt_dens_device[nijk_gas]*sqrt_dens_device[nijk_gas]*denskr_device[nijk_gas];
            }
		}
	}
}





__global__
void cal_grad(int *na_device, int *nb_device, int *nc_device, double *chem_device, hipfftDoubleComplex *FFT_out_device, 
	double *denskr_device, double *Vext_device, double *sqrt_dens_device, double *rho_bulk_device, double *gradient_device, int j)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int i;
	int batch_i;
	for (i=index; i<(na_device[0]*nb_device[0]*nc_device[0]); i+=stride)
	{
		batch_i = i + j*na_device[0]*nb_device[0]*nc_device[0];

		if ((sqrt_dens_device[batch_i]*sqrt_dens_device[batch_i])>(1.0e-5*rho_bulk_device[j]))
		{
			gradient_device[batch_i] = log(sqrt_dens_device[batch_i]*sqrt_dens_device[batch_i]) + Vext_device[batch_i] - chem_device[j] + FFT_out_device[i].x + denskr_device[batch_i];
		}
		else
		{
			gradient_device[batch_i] = 0;
		}
		gradient_device[batch_i] = gradient_device[batch_i]*2*sqrt_dens_device[batch_i];
	}
}





double myvalue(double *sqrt_dens, long int system_size)
{
	hipMemcpy(sqrt_dens_device, sqrt_dens, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyHostToDevice);
	// 
	int i, j;



	// initialize the weighted density for hard sphere term
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n0_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n1_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n2_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n3_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1z_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2z_FFT_in_device);

	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denskr_device);
	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denski_device);

	// calculate the weighted density
    for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], sqrt_dens_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Forward FFT of density FAILED at specie:\n");
		}

		add_weighted_density<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na_device, nb_device, nc_device, diameter_HS_device, k_gas_device, FFT_out_device, n0_FFT_in_device, 
		n1_FFT_in_device, n2_FFT_in_device, n3_FFT_in_device, nv1x_FFT_in_device, nv1y_FFT_in_device, nv1z_FFT_in_device, 
		nv2x_FFT_in_device, nv2y_FFT_in_device, nv2z_FFT_in_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, 
		w_nv2y_FFT_device, w_nv2z_FFT_device, uatt_device, denskr_device, denski_device, j);
    }

    hipfftExecZ2Z(fft_plan_device, n0_FFT_in_device, n0_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n1_FFT_in_device, n1_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n2_FFT_in_device, n2_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n3_FFT_in_device, n3_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1x_FFT_in_device, nv1x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1y_FFT_in_device, nv1y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1z_FFT_in_device, nv1z_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2x_FFT_in_device, nv2x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2y_FFT_in_device, nv2y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2z_FFT_in_device, nv2z_FFT_out_device, HIPFFT_BACKWARD);

	for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Backward FFT of density FAILED at specie:\n");
		}

		complext_split_into_real_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_out_device, j);
    }

    // calculate grand potential
    cal_grand_potential<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, k_gas_device, Vext_device, chem_device, sqrt_dens_device, rho_bulk_device, n0_FFT_out_device, 
	n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, 
	nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device, denskr_device, grand_pontential_device);

    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
	hipMemcpy(grand_pontential, grand_pontential_device, sizeof(double)*na[0]*nb[0]*nc[0], hipMemcpyDeviceToHost);
	double result = 0;
	for (i=0; i<na[0]*nb[0]*nc[0]; i++)
	{
		result += grand_pontential[i];
	}
	result = result * (la_expanded[0]/(na[0])) * (lb_expanded[0]/(nb[0])) * (lc_expanded[0]/(nc[0]));

	// printf("free energy: %lf\n", result);

	return result;
}





void mygrad(double *gradient, double *sqrt_dens, long int system_size)
{
	hipMemcpy(sqrt_dens_device, sqrt_dens, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyHostToDevice);
	// 
	int i, j;



	// initialize the weighted density for hard sphere term
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n0_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n1_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n2_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n3_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1z_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2z_FFT_in_device);

	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denskr_device);
	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denski_device);

	// calculate the weighted density
    for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], sqrt_dens_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Forward FFT of density FAILED at specie:\n");
		}

		add_weighted_density<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na_device, nb_device, nc_device, diameter_HS_device, k_gas_device, FFT_out_device, n0_FFT_in_device, 
		n1_FFT_in_device, n2_FFT_in_device, n3_FFT_in_device, nv1x_FFT_in_device, nv1y_FFT_in_device, nv1z_FFT_in_device, 
		nv2x_FFT_in_device, nv2y_FFT_in_device, nv2z_FFT_in_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, 
		w_nv2y_FFT_device, w_nv2z_FFT_device, uatt_device, denskr_device, denski_device, j);
    }

    hipfftExecZ2Z(fft_plan_device, n0_FFT_in_device, n0_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n1_FFT_in_device, n1_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n2_FFT_in_device, n2_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n3_FFT_in_device, n3_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1x_FFT_in_device, nv1x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1y_FFT_in_device, nv1y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1z_FFT_in_device, nv1z_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2x_FFT_in_device, nv2x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2y_FFT_in_device, nv2y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2z_FFT_in_device, nv2z_FFT_out_device, HIPFFT_BACKWARD);

	for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Backward FFT of density FAILED at specie:\n");
		}

		complext_split_into_real_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_out_device, j);
    }

    //calculate the derivative term of weighted density
    cal_F_deri<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, n0_FFT_out_device, n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, 
	nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device,
	n0_FFT_in_device, n1_FFT_in_device, n2_FFT_in_device, n3_FFT_in_device, nv1x_FFT_in_device, nv1y_FFT_in_device, nv1z_FFT_in_device, 
	nv2x_FFT_in_device, nv2y_FFT_in_device, nv2z_FFT_in_device);



	hipfftExecZ2Z(fft_plan_device, n0_FFT_in_device, n0_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n1_FFT_in_device, n1_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n2_FFT_in_device, n2_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n3_FFT_in_device, n3_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1x_FFT_in_device, nv1x_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1y_FFT_in_device, nv1y_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1z_FFT_in_device, nv1z_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2x_FFT_in_device, nv2x_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2y_FFT_in_device, nv2y_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2z_FFT_in_device, nv2z_FFT_out_device, HIPFFT_FORWARD);



	for (j=0; j<k_gas[0]; j++)
    {
    	sum_F_deri<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na_device, nb_device, nc_device, diameter_HS_device, k_gas_device, FFT_in_device, n0_FFT_out_device, 
		n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, 
		nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, 
		w_nv2y_FFT_device, w_nv2z_FFT_device, j);

		if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Forward FFT of density FAILED at specie:\t%d\n", i);
		}

		cal_grad<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na_device, nb_device, nc_device, chem_device, FFT_out_device, denskr_device, Vext_device, sqrt_dens_device, rho_bulk_device, gradient_device, j);
    }

    hipMemcpy(gradient, gradient_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyDeviceToHost);
}





double myvalgrad(double *gradient, double *sqrt_dens, long int system_size)
{
	hipMemcpy(sqrt_dens_device, sqrt_dens, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyHostToDevice);
	// 
	int i, j;



	// initialize the weighted density for hard sphere term
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n0_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n1_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n2_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], n3_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv1z_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2x_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2y_FFT_in_device);
	ini_device_value_complex<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0], nv2z_FFT_in_device);

	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denskr_device);
	ini_device_value_double<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
	(na[0]*nb[0]*nc[0]*k_gas[0], denski_device);

	// calculate the weighted density
    for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], sqrt_dens_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Forward FFT of density FAILED at specie:\n");
		}

		add_weighted_density<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na_device, nb_device, nc_device, diameter_HS_device, k_gas_device, FFT_out_device, n0_FFT_in_device, 
		n1_FFT_in_device, n2_FFT_in_device, n3_FFT_in_device, nv1x_FFT_in_device, nv1y_FFT_in_device, nv1z_FFT_in_device, 
		nv2x_FFT_in_device, nv2y_FFT_in_device, nv2z_FFT_in_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, 
		w_nv2y_FFT_device, w_nv2z_FFT_device, uatt_device, denskr_device, denski_device, j);

    }

    hipfftExecZ2Z(fft_plan_device, n0_FFT_in_device, n0_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n1_FFT_in_device, n1_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n2_FFT_in_device, n2_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, n3_FFT_in_device, n3_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1x_FFT_in_device, nv1x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1y_FFT_in_device, nv1y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv1z_FFT_in_device, nv1z_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2x_FFT_in_device, nv2x_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2y_FFT_in_device, nv2y_FFT_out_device, HIPFFT_BACKWARD);
	hipfftExecZ2Z(fft_plan_device, nv2z_FFT_in_device, nv2z_FFT_out_device, HIPFFT_BACKWARD);

	for (j=0; j<k_gas[0]; j++)
    {
    	real_split_into_complex_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_in_device, j);

    	if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Backward FFT of density FAILED at specie:\n");
		}

		complext_split_into_real_separate_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na[0]*nb[0]*nc[0], denskr_device, denski_device, FFT_out_device, j);
    }

    // calculate grand potential
    cal_grand_potential<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, k_gas_device, Vext_device, chem_device, sqrt_dens_device, rho_bulk_device, n0_FFT_out_device, 
	n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, 
	nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device, denskr_device, grand_pontential_device);

    // calculate the derivative term of weighted density
    cal_F_deri<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, n0_FFT_out_device, n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, 
	nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device,
	n0_FFT_in_device, n1_FFT_in_device, n2_FFT_in_device, n3_FFT_in_device, nv1x_FFT_in_device, nv1y_FFT_in_device, nv1z_FFT_in_device, 
	nv2x_FFT_in_device, nv2y_FFT_in_device, nv2z_FFT_in_device);

	//Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
    //Revision needed for the speed!!!!!!!!!
	hipMemcpy(grand_pontential, grand_pontential_device, sizeof(double)*na[0]*nb[0]*nc[0], hipMemcpyDeviceToHost);
	double result = 0;
	for (i=0; i<na[0]*nb[0]*nc[0]; i++)
	{
		result += grand_pontential[i];
	}
	result = result * (la_expanded[0]/(na[0])) * (lb_expanded[0]/(nb[0])) * (lc_expanded[0]/(nc[0]));

	hipfftExecZ2Z(fft_plan_device, n0_FFT_in_device, n0_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n1_FFT_in_device, n1_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n2_FFT_in_device, n2_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, n3_FFT_in_device, n3_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1x_FFT_in_device, nv1x_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1y_FFT_in_device, nv1y_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv1z_FFT_in_device, nv1z_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2x_FFT_in_device, nv2x_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2y_FFT_in_device, nv2y_FFT_out_device, HIPFFT_FORWARD);
	hipfftExecZ2Z(fft_plan_device, nv2z_FFT_in_device, nv2z_FFT_out_device, HIPFFT_FORWARD);
	
	for (j=0; j<k_gas[0]; j++)
    {
    	sum_F_deri<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
    	(na_device, nb_device, nc_device, diameter_HS_device, k_gas_device, FFT_in_device, n0_FFT_out_device, 
		n1_FFT_out_device, n2_FFT_out_device, n3_FFT_out_device, nv1x_FFT_out_device, nv1y_FFT_out_device, nv1z_FFT_out_device, 
		nv2x_FFT_out_device, nv2y_FFT_out_device, nv2z_FFT_out_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, 
		w_nv2y_FFT_device, w_nv2z_FFT_device, j);

		if (hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("CUFFT error: Forward FFT of density FAILED at specie:\t%d\n", i);
		}

		cal_grad<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		(na_device, nb_device, nc_device, chem_device, FFT_out_device, denskr_device, Vext_device, sqrt_dens_device, rho_bulk_device, gradient_device, j);
    }

    hipMemcpy(gradient, gradient_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyDeviceToHost);
    return result;
}























int main(int argc, char *argv[])
{
	clock_t t;
	t = clock();
	clock_t temp_t;
	double t_ini, t_w_d=0, t_f_deri=0;
	// clock_t t_ini, t_w_d, t_f_deri;

	// define variables used for file I/O
	FILE *fp1;
    int buffersize = 256;
    char str[buffersize];
    int int_a;
    // define variables assisst the main fxn
    int i, ii, iii, iiii;
    int j, jj;
    
    
    
    // define key variables read from the input file and independent of kind of gas
    int *Nmax_a, *Nmax_b, *Nmax_c;
    double *la, *lb, *lc, *dl;
    double *alpha, *beta, *gamma;
    double *temperature;
    int *signal_read_Vext, *signal_EOS;
    double *cutoff, *cutoff_device;
    int *N_atoms;
    // allocate memory for those variables
    Nmax_a = (int *) malloc(sizeof(int));
    Nmax_b = (int *) malloc(sizeof(int));
    Nmax_c = (int *) malloc(sizeof(int));
    la = (double *) malloc(sizeof(double));
    lb = (double *) malloc(sizeof(double));
    lc = (double *) malloc(sizeof(double));
    dl = (double *) malloc(sizeof(double));
    alpha = (double *) malloc(sizeof(double));
    beta = (double *) malloc(sizeof(double));
    gamma = (double *) malloc(sizeof(double));
    temperature = (double *) malloc(sizeof(double));
    signal_read_Vext = (int *) malloc(sizeof(int));
    signal_EOS = (int *) malloc(sizeof(int));
    N_atoms = (int *) malloc(sizeof(int));
    // allocate pinned memory
    hipHostMalloc(&k_gas, sizeof(int));
    hipHostMalloc(&cutoff, sizeof(double));
    // allocate GPU memory
    hipMalloc((void **)&k_gas_device, sizeof(int)*1);
    hipMalloc((void **)&cutoff_device, sizeof(double)*1);
    // read varaiables from input
    fp1 = fopen(argv[1], "r");
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%d %d %d\n", &Nmax_a[0], &Nmax_b[0], &Nmax_c[0]);
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%lf %lf %lf %lf\n", &la[0], &lb[0], &lc[0], &dl[0]);
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%lf %lf %lf\n", &alpha[0], &beta[0], &gamma[0]);
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%lf ", &temperature[0]);
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%d\n", &k_gas[0]);
    hipMemcpy(k_gas_device, k_gas, sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(cutoff_device, cutoff, sizeof(double), hipMemcpyHostToDevice);
    // check_int<<<1,32>>>(1, k_gas_device);
    // check_double<<<1,32>>>(1, cutoff_device);
    

    
    // define variables dependent on the kind of gas
    int *index_gas;
    double *epsilon;
    double *epsilon_star, *epsilon_star_device;
    double *sigma, *sigma_device;
    double *rho_bulk;
    // allocate memory for the variables
    index_gas = (int *) malloc(sizeof(int)*k_gas[0]);
    epsilon = (double *) malloc(sizeof(double)*k_gas[0]);
    epsilon_star = (double *) malloc(sizeof(double)*k_gas[0]);
    sigma = (double *) malloc(sizeof(double)*k_gas[0]);
    rho_bulk = (double *) malloc(sizeof(double)*k_gas[0]);
    // allocate pinned memory
    hipHostMalloc(&epsilon_star, sizeof(double)*k_gas[0]);
    hipHostMalloc(&sigma, sizeof(double)*k_gas[0]);
    hipHostMalloc(&rho_bulk, sizeof(double)*k_gas[0]);
    // allocate GPU memory
    hipMalloc((void **)&epsilon_star_device, sizeof(double)*k_gas[0]);
    hipMalloc((void **)&sigma_device, sizeof(double)*k_gas[0]);
    hipMalloc((void **)&rho_bulk_device, sizeof(double)*k_gas[0]);

    // read variables from input
    fgets(str, buffersize, fp1);
    for (i=0; i<k_gas[0]; i++)
    {
        fscanf(fp1, "%d %lf %lf %lf\n", &index_gas[i], &epsilon[i], &sigma[i], &rho_bulk[i]);
        epsilon_star[i] = 1.0*epsilon[i]/temperature[0];
    }
    hipMemcpy(epsilon_star_device, epsilon_star, sizeof(double)*k_gas[0], hipMemcpyHostToDevice);
    hipMemcpy(sigma_device, sigma, sizeof(double)*k_gas[0], hipMemcpyHostToDevice);
    hipMemcpy(rho_bulk_device, rho_bulk, sizeof(double)*k_gas[0], hipMemcpyHostToDevice);


    // check_double<<<1,32>>>(k_gas[0], epsilon_star_device);
    // check_double<<<1,32>>>(k_gas[0], sigma_device);
    // check_double<<<1,32>>>(k_gas[0], rho_bulk_device);
    
    
    
    // skip special case!!!!!!!!!!!!!
    // skip special case!!!!!!!!!!!!!
    // skip special case!!!!!!!!!!!!!
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    
    
    
    // read other variables
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%d %d %lf", &signal_read_Vext[0], &signal_EOS[0], &cutoff[0]);
    hipMemcpy(cutoff_device, cutoff, sizeof(double)*1, hipMemcpyHostToDevice);
    // check_double<<<1,32>>>(1, cutoff_device);
    fgets(str, buffersize, fp1);
    
    
    
    // skip excess entropy scalling parameters!!!!!!!!!!!!!!!!!
    // skip excess entropy scalling parameters!!!!!!!!!!!!!!!!!
    // skip excess entropy scalling parameters!!!!!!!!!!!!!!!!!
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    
    
    
    // read host materials
    fgets(str, buffersize, fp1);
    fscanf(fp1, "%d\n", &N_atoms[0]);
    
    // creat variables based on input files
    double *sigma_host, *epsilon_host;
    double *x_host, *y_host, *z_host;
    // allocate memory
    sigma_host = (double *) malloc(N_atoms[0]*sizeof(double));
    epsilon_host = (double *) malloc(N_atoms[0]*sizeof(double));
    x_host = (double *) malloc(N_atoms[0]*sizeof(double));
    y_host = (double *) malloc(N_atoms[0]*sizeof(double));
    z_host = (double *) malloc(N_atoms[0]*sizeof(double));
    
    // continue read info from input file
    fgets(str, buffersize, fp1);
    for (i=0; i<N_atoms[0]; i++)
    {
        fscanf(fp1,"%d %lf %lf\n", &int_a, &sigma_host[i], &epsilon_host[i]);
    }
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    fgets(str, buffersize, fp1);
    for (i=0; i<N_atoms[0]; i++)
    {
        fscanf(fp1,"%d %lf %lf %lf\n", &int_a, &x_host[i], &y_host[i], &z_host[i]);
    }
    fclose(fp1);
    
    
    
    // check whether input parameters are correct
    // printf("1:\t%d\t%d\t%d\n", Nmax_a[0], Nmax_b[0], Nmax_c[0]);
    // printf("2:\t%lf\t%lf\t%lf\t%lf\n", la[0], lb[0], lc[0], dl[0]);
    // printf("3:\t%lf\t%lf\t%lf\n", alpha[0], beta[0], gamma[0]);
    // printf("4:\t%lf\n", temperature[0]);
    // printf("5:\t%d\n", k_gas[0]);
    // for (i=0; i<k_gas[0]; i++)
    // {
    //     printf("index of gas:\t%d\t%lf\t%lf\t%lf\n", index_gas[i], epsilon[i], sigma[i], rho_bulk[i]);
    // }
    // printf("6:\t%d\t%d\t%lf\n", signal_read_Vext[0], signal_EOS[0], cutoff[0]);
    // printf("7:\t%d\n", N_atoms[0]);
    // for (i=0; i<N_atoms[0]; i++)
    // {
    //     printf("%d %lf %lf\n", i+1, sigma_host[i], epsilon_host[i]);
    // }
    // for (i=0; i<N_atoms[0]; i++)
    // {
    //     printf("%d %lf %lf %lf\n", i+1, x_host[i], y_host[i], z_host[i]);
    // }


    


    // The following section is correct assume the lattice cell is cubic.
    // Since Yu's program is not incorrect for triclinic cell, we are only
    // making sure the corresponding C code work for the cubic case

    // define variables used for expanding original to satisfy the periodic boundary condition
    int *expand_times_a, *expand_times_b, *expand_times_c, *expand_times_total;
    double *la_expanded_device;
    double *lb_expanded_device;
    double *lc_expanded_device;
    
    int *N_atoms_expanded, *N_atoms_expanded_device;
    // allocate memory
    expand_times_a = (int *) malloc(sizeof(int));
    expand_times_b = (int *) malloc(sizeof(int));
    expand_times_c = (int *) malloc(sizeof(int));
    expand_times_total = (int *) malloc(sizeof(int));
    // allocate pinned memory
    hipHostMalloc(&la_expanded, sizeof(double));
    hipHostMalloc(&lb_expanded, sizeof(double));
    hipHostMalloc(&lc_expanded, sizeof(double));
    hipHostMalloc(&na, sizeof(int));
    hipHostMalloc(&nb, sizeof(int));
    hipHostMalloc(&nc, sizeof(int));
    hipHostMalloc(&N_atoms_expanded, sizeof(int));
    // allocate GPU memory
    hipMalloc((void **)&la_expanded_device, sizeof(double)*1);
    hipMalloc((void **)&lb_expanded_device, sizeof(double)*1);
    hipMalloc((void **)&lc_expanded_device, sizeof(double)*1);
    hipMalloc((void **)&na_device, sizeof(int)*1);
    hipMalloc((void **)&nb_device, sizeof(int)*1);
    hipMalloc((void **)&nc_device, sizeof(int)*1);
    hipMalloc((void **)&N_atoms_expanded_device, sizeof(int)*1);

    // calculate expanding time in each axis
    expand_times_a[0] = (int) ((2*cutoff[0]/la[0]) + 1);
    expand_times_b[0] = (int) ((2*cutoff[0]/lb[0]) + 1);
    expand_times_c[0] = (int) ((2*cutoff[0]/lc[0]) + 1);
    expand_times_total[0] = (int) (expand_times_a[0]*expand_times_b[0]*expand_times_c[0]);
    N_atoms_expanded[0] = N_atoms[0] * expand_times_total[0];
    la_expanded[0] = la[0] * expand_times_a[0];
    lb_expanded[0] = lb[0] * expand_times_b[0];
    lc_expanded[0] = lc[0] * expand_times_c[0];
    na[0] = (int) (la_expanded[0]/dl[0] + 1);
    if (na[0] > Nmax_a[0])
    {
        na[0] = Nmax_a[0];
    }
    nb[0] = (int) (lb_expanded[0]/dl[0] + 1);
    if (nb[0] > Nmax_b[0])
    {
        nb[0] = Nmax_b[0];
    }
    nc[0] = (int) (lc_expanded[0]/dl[0] + 1);
    if (nc[0] > Nmax_c[0])
    {
        nc[0] = Nmax_c[0];
    }
    hipMemcpy(na_device, na, sizeof(int)*1, hipMemcpyHostToDevice);
    hipMemcpy(nb_device, nb, sizeof(int)*1, hipMemcpyHostToDevice);
    hipMemcpy(nc_device, nc, sizeof(int)*1, hipMemcpyHostToDevice);
    hipMemcpy(la_expanded_device, la_expanded, sizeof(double)*1, hipMemcpyHostToDevice);
    hipMemcpy(lb_expanded_device, lb_expanded, sizeof(double)*1, hipMemcpyHostToDevice);
    hipMemcpy(lc_expanded_device, lc_expanded, sizeof(double)*1, hipMemcpyHostToDevice);
    hipMemcpy(N_atoms_expanded_device, N_atoms_expanded, sizeof(int)*1, hipMemcpyHostToDevice);
    // check_int<<<1,32>>>(1, na_device);
    // check_int<<<1,32>>>(1, nb_device);
    // check_int<<<1,32>>>(1, nc_device);
    // check_double<<<1,32>>>(1, la_expanded_device);
    // check_double<<<1,32>>>(1, lb_expanded_device);
    // check_double<<<1,32>>>(1, lc_expanded_device);
    // check_int<<<1,32>>>(1, N_atoms_expanded_device);

    // define variables dependent on the expand times in each axis
    double *sigma_host_expanded, *sigma_host_expanded_device;
    double *epsilon_host_star_expanded, *epsilon_host_star_expanded_device;
    double *x_host_expanded, *x_host_expanded_device;
    double *y_host_expanded, *y_host_expanded_device;
    double *z_host_expanded, *z_host_expanded_device;
    // allocate pinned memroy
    hipHostMalloc(&sigma_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipHostMalloc(&epsilon_host_star_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipHostMalloc(&x_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipHostMalloc(&y_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipHostMalloc(&z_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    // allocate GPU memory
    hipMalloc((void **)&sigma_host_expanded_device, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipMalloc((void **)&epsilon_host_star_expanded_device, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipMalloc((void **)&x_host_expanded_device, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipMalloc((void **)&y_host_expanded_device, sizeof(double)*expand_times_total[0]*N_atoms[0]);
    hipMalloc((void **)&z_host_expanded_device, sizeof(double)*expand_times_total[0]*N_atoms[0]);


    
    j = 0;
    for (i=0; i<N_atoms[0]; i++)
    {
        for (ii=0; ii<expand_times_a[0]; ii++)
        {
            for (iii=0; iii<expand_times_b[0]; iii++)
            {
                for (iiii=0; iiii<expand_times_c[0]; iiii++)
                {
                    epsilon_host_star_expanded[j] = 1.0*epsilon_host[i]/temperature[0];
                    sigma_host_expanded[j] = sigma_host[i];
                    x_host_expanded[j] = x_host[i] + ii*la[0];
                    y_host_expanded[j] = y_host[i] + iii*lb[0];
                    z_host_expanded[j] = z_host[i] + iiii*lc[0];
                    j++;
                }
            }
        }
    }
    hipMemcpy(sigma_host_expanded_device, sigma_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0], hipMemcpyHostToDevice);
    hipMemcpy(epsilon_host_star_expanded_device, epsilon_host_star_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0], hipMemcpyHostToDevice);
    hipMemcpy(x_host_expanded_device, x_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0], hipMemcpyHostToDevice);
    hipMemcpy(y_host_expanded_device, y_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0], hipMemcpyHostToDevice);
    hipMemcpy(z_host_expanded_device, z_host_expanded, sizeof(double)*expand_times_total[0]*N_atoms[0], hipMemcpyHostToDevice);
    // check_double<<<1,32>>>(expand_times_total[0]*N_atoms[0], sigma_host_expanded_device);
    // check_double<<<1,32>>>(expand_times_total[0]*N_atoms[0], epsilon_host_star_expanded);
    // check_double<<<1,32>>>(expand_times_total[0]*N_atoms[0], x_host_expanded_device);
    // check_double<<<1,32>>>(expand_times_total[0]*N_atoms[0], y_host_expanded_device);
    // check_double<<<1,32>>>(expand_times_total[0]*N_atoms[0], z_host_expanded_device);
    if (j!=N_atoms_expanded[0])
    {
        printf("error in expanding the original cell!!!\n");
    }
    // printf("expand_time:\t%d\t%d\t%d\n", expand_times_a[0], expand_times_b[0], expand_times_c[0]);
    // printf("li_expanded:\t%lf\t%lf\t%lf\n", la_expanded[0], lb_expanded[0], lc_expanded[0]);
    // printf("li_expanded:\t%d\t%d\t%d\n", na[0], nb[0], nc[0]);

    // free the memory occupied by the original cell
    free(epsilon_host);
    free(sigma_host);
    free(x_host);
    free(y_host);
    free(z_host);
    free(la);
    free(lb);
    free(lc);
    free(N_atoms);





    // calculate corresponding hard sphere diameter according to LJ parameters
    double *Tstar;
    double *diameter_HS;
    Tstar = (double *) malloc(sizeof(double)*k_gas[0]);
    hipHostMalloc(&diameter_HS, sizeof(double)*k_gas[0]);
    hipMalloc((void **)&diameter_HS_device, sizeof(double)*k_gas[0]);

    for (i=0; i<k_gas[0]; i++)
    {
        Tstar[i] = temperature[0]/epsilon[i];
        if ((Tstar[i]>0) && (Tstar[i]<15))
        {
            // printf("Reduced temperature is within good range.\n");
        }
        else
        {
            // printf("WARNING: reduced temperature is out of reasonable range!!!!!!\n");
            // printf("Index of gas: %d\tReduced Temperature: %lf\n", i, Tstar[i]);
        }
        diameter_HS[i] = (1+0.2977*Tstar[i]) / (1+0.33163*Tstar[i]+0.0010477*pow(Tstar[i],2))*sigma[i];
        // printf("%lf\t%lf\n", Tstar[i], diameter_HS[i]);
    }
    hipMemcpy(diameter_HS_device, diameter_HS, sizeof(double)*k_gas[0], hipMemcpyHostToDevice);
    // check_double<<<1,32>>>(k_gas[0], diameter_HS_device);





    // variables for multifunction used in the initiliazatiion
    int *nijk_gas;
    nijk_gas = (int *) malloc(sizeof(int));


    
    // allocate GPU memory
    hipMalloc((void **)&w_n0_FFT_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&w_n3_FFT_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&w_nv2x_FFT_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&w_nv2y_FFT_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&w_nv2z_FFT_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&uatt_device, sizeof(double)*na[0]*nb[0]*nc[0]*(k_gas[0] + (int) (0.5*(k_gas[0]*(k_gas[0]-1)))));



    // This sectin works the same as the interpolation way Yu Liu has for the weight density but with GPU
    // However, this following section follows a more standard routine
    cal_weight_density<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, la_expanded_device, lb_expanded_device, lc_expanded_device, k_gas_device, 
    diameter_HS_device, w_n0_FFT_device, w_n3_FFT_device, w_nv2x_FFT_device, w_nv2y_FFT_device, w_nv2z_FFT_device);


	
    hipMalloc((void **)&FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    
    if (hipfftPlan3d(&fft_plan_device, na[0], nb[0], nc[0], HIPFFT_Z2Z) != HIPFFT_SUCCESS)
	{
		printf("CUFFT error: Plan creation failed\n");
	}




    
    // putting attraction terms on GPU in order to speed up GPU program
    // and also to avoid the call of fftw3
    for (j=0; j<k_gas[0]; j++)
    {
    	for (jj=0; jj<=j; jj++)
    	{
    		cal_lj_pre_freq_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		    (na_device, nb_device, nc_device, la_expanded_device, lb_expanded_device, lc_expanded_device, sigma_device, 
		    epsilon_star_device, cutoff_device, j, jj, FFT_in_device);

		    hipfftExecZ2Z(fft_plan_device, FFT_in_device, FFT_out_device, HIPFFT_FORWARD);

		    cal_lj_freq_store_batch<<<(int)((na[0]*nb[0]*nc[0]-1)/running_block_size+1),running_block_size>>>
		    (na_device, nb_device, nc_device, la_expanded_device, lb_expanded_device, lc_expanded_device, j, jj, 
		    FFT_out_device, uatt_device);


    	}
    }





    // calculate the residual chemical potential and full chemical potential
    double *chem;
    double *bulk_pressure;
    bulk_pressure = (double *) malloc(sizeof(double));
    // allocate pinned memory
    hipHostMalloc(&chem, sizeof(double)*k_gas[0]);
    // allocate GPU memory
    hipMalloc((void **)&chem_device, sizeof(double)*k_gas[0]);

    // calculate the system pressure and reduced chemical potential
    new_MBWR(k_gas, epsilon, sigma, temperature, rho_bulk, chem, bulk_pressure);
    
    hipMemcpy(chem_device, chem, sizeof(double)*k_gas[0], hipMemcpyHostToDevice);
    // printf("chemical potential\n");
    // check_double<<<1,32>>>(k_gas[0], chem_device);





    // define variables
    double *sqrt_dens;
    // allocate pinned memroy
    hipHostMalloc(&sqrt_dens, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    // allocate GPU memory
    hipMalloc((void **)&sqrt_dens_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);


    
    hipMalloc((void **)&Vext_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);



    cal_Vext<<<(int)((na[0]*nb[0]*nc[0]*k_gas[0]-1)/running_block_size+1),running_block_size>>>
    (na_device, nb_device, nc_device, la_expanded_device, lb_expanded_device, lc_expanded_device, 
	N_atoms_expanded_device, epsilon_host_star_expanded_device, sigma_host_expanded_device,
	x_host_expanded_device, y_host_expanded_device, z_host_expanded_device,
	k_gas_device, epsilon_star_device, sigma_device, rho_bulk_device, 
	diameter_HS_device, sqrt_dens_device, cutoff_device, Vext_device);



    hipMemcpy(sqrt_dens, sqrt_dens_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0], hipMemcpyDeviceToHost);


	
    hipMalloc((void **)&n0_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n1_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n2_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n3_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1x_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1y_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1z_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2x_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2y_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2z_FFT_in_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);

    
    hipMalloc((void **)&n0_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n1_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n2_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&n3_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1x_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1y_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv1z_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2x_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2y_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);
    hipMalloc((void **)&nv2z_FFT_out_device, sizeof(hipfftDoubleComplex)*na[0]*nb[0]*nc[0]);



    
    hipMalloc((void **)&denskr_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    hipMalloc((void **)&denski_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);



    double *tot_density;
    tot_density = (double *) malloc(sizeof(double)*k_gas[0]);



    // allocate pinned memory
    hipHostMalloc(&grand_pontential, sizeof(double)*na[0]*nb[0]*nc[0]);
    // allocate GPU memory
    hipMalloc((void **)&grand_pontential_device, sizeof(double)*na[0]*nb[0]*nc[0]);

    int system_size = na[0]*nb[0]*nc[0]*k_gas[0];
    
    double *gradient;
    // allocate pinned memory
    hipHostMalloc(&gradient, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);
    // allocate GPU memory
    hipMalloc((void **)&gradient_device, sizeof(double)*na[0]*nb[0]*nc[0]*k_gas[0]);




    // myvalue(sqrt_dens, system_size);
    // check_double<<<1,32>>>(k_gas[0], rho_bulk_device);
    // mygrad(gradient, sqrt_dens, system_size);
    // myvalgrad(gradient, sqrt_dens, system_size);
    
    // hipDeviceSynchronize();
    // return 0;


    int output_signal;
    output_signal = cg_descent(sqrt_dens, system_size, NULL, NULL, 1.e-3, myvalue, mygrad, myvalgrad, NULL) ;
    hipDeviceSynchronize();
    // printf("%s\t", argv[1]);
    printf("%d\t", output_signal);
    // return 0;

    t = clock() - t;
    double time_used;
    time_used = ((double)t)/CLOCKS_PER_SEC;
    printf("%lf\t", bulk_pressure[0]);
    for (j=0; j<k_gas[0]; j++)
	{
		tot_density[j] = 0;
		for (i=0; i<na[0]*nb[0]*nc[0]; i++)
		{
			nijk_gas[0] = i + j*na[0]*nb[0]*nc[0];
			// dens[nijk_gas[0]] = dens_new[nijk_gas[0]]*mix_F[0] + (1-mix_F[0])*dens[nijk_gas[0]];
			tot_density[j] = tot_density[j] + sqrt_dens[nijk_gas[0]]*sqrt_dens[nijk_gas[0]];
		}
		tot_density[j] = 1.0*tot_density[j]*1.0e4/6.023/na[0]/nb[0]/nc[0];
		// tot_density[j] = 1.0*tot_density[j]/na[0]/nb[0]/nc[0];
		// printf("adsorption amount: %lf (mol/L)\n", tot_density[j]);
		printf("%.5e\t", tot_density[j]);
	}
	printf("\n");
}
